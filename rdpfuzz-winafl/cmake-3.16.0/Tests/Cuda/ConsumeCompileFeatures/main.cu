

#include <hip/hip_runtime.h>
#include <iostream>

int static_cxx11_func(int);

void test_functions()
{
  auto x = static_cxx11_func(int(42));
  std::cout << x << std::endl;
}

int main(int argc, char** argv)
{
  test_functions();
  std::cout
    << "this executable doesn't use cuda code, just call methods defined"
    << std::endl;
  std::cout << "in libraries that have cuda code" << std::endl;
  return 0;
}
