#include "hip/hip_runtime.h"


#include "file1.h"
#include "file2.h"

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

static __global__ void file3_kernel(result_type& r, int x)
{
  // call static_func which is a method that is defined in the
  // static library that is always out of date
  r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

result_type file3_launch_kernel(int x)
{
  result_type r;
  file3_kernel<<<1, 1>>>(r, x);
  return r;
}
